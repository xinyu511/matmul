// nvcc matmul_tmem.cu -std=c++20 -O3 -arch=sm_100a -Xcompiler -fopenmp -o matmul_tmem [-DDEBUG_UMMA]
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <cstdint>
#include <vector>
#include <random>
#include <cstdio>
#include <cmath>
#include <chrono>
#include <iostream>
#include <omp.h>
#include <fstream>

constexpr int Mb = 128, Nb = 128, Kb = 64;
using  Tab  = half;
using  Tacc = float;



#define uint unsigned int
#define uchar unsigned char
#define ushort unsigned short
#define int64_t long long
#define uint64_t unsigned long long

//copied from thunderkittens
template<int phaseBit>
__device__ inline void mbarrier_wait(void* bar_b64)
{
    uint32_t addr = static_cast<uint32_t>(__cvta_generic_to_shared(bar_b64));
    asm volatile (
        "{\n"
        ".reg .pred                P1;\n"
        "LAB_WAIT:\n"
        "mbarrier.try_wait.parity.acquire.cluster.shared::cta.b64 P1, [%0], %1;\n"
        "@P1                       bra.uni DONE;\n"
        "bra.uni                   LAB_WAIT;\n"
        "DONE:\n"
        "}\n"
        :: "r"(addr),
        "r"(phaseBit)
    );
}


__forceinline__ __device__ uint32_t get_tmem_addr(uint32_t idx, int row_offset, int col_offset) {
  int col_idx = idx & 0xFFFF;
  int row_idx = (idx >> 16) & 0xFFFF;
  col_idx += col_offset;
  row_idx += row_offset;
  col_idx = col_idx & 0xFFFF;
  row_idx = row_idx & 0xFFFF;

  uint32_t new_idx = (row_idx << 16) | col_idx;
  return new_idx;
}

#ifndef HOST_DEVICE
#define HOST_DEVICE __forceinline__ __host__ __device__
#endif
union SmemDescriptor
{
  uint64_t desc_ = 0;
  // Bitfield implementation avoids the need for shifts in assignment
  struct {
    // start_address, bit [0,14), 4LSB not included
    uint16_t start_address_ : 14, : 2;                     // 14 bits [0,14), 2 bits unused
    // leading dimension byte offset, bit [16,30), 4LSB not included
    uint16_t leading_byte_offset_ : 14, : 2;               // 14 bits [0,14), 2 bits unused
    // stride dimension byte offset, bit [32,46), 4LSB not included
    uint16_t stride_byte_offset_ : 14, version_ : 2;       // 14 bits [0,14), 2 bits [14,16)
    // base_offset, bit [49,52). leading_byte_offset_mode, bit [52,53).
    uint8_t : 1, base_offset_ : 3, lbo_mode_ : 1, : 3;     // 1 bit unused, 3 bits [1,4), 1 bit [4,5), 3 bits unused
    // layout type, bit [61,64), SWIZZLE_NONE matrix descriptor = 0, SWIZZLE_128B matrix descriptor = 2, SWIZZLE_64B descriptor = 4, SWIZZLE_32B descriptor = 6, SWIZZLE_128B_BASE32B = 1, N/A = 3, N/A = 5, N/A = 7
    uint8_t : 5, layout_type_ : 3;                         // 6 bits unused, 3 bits [5,8)
  };
  // Seperate the field, as we may only update one part of desc
  struct {
    uint32_t lo;
    uint32_t hi;
  };

  // Decay to a uint64_t
  HOST_DEVICE constexpr
  operator uint64_t() const noexcept { return desc_; }
};

#ifndef HOST_DEVICE
#define HOST_DEVICE __forceinline__ __host__ __device__
#endif
union InstrDescriptor
{
  uint32_t desc_;

  struct {
    // Bitfield implementation avoids the need for shifts in assignment
    uint16_t sparse_id2_    : 2,  // bit [ 0, 2) : Sparse meta data id2
             sparse_flag_   : 1,  // bit [ 2, 3) : 0 = dense. 1 = sparse. 1 value valid only for F32F16/S8/MXF8F6F4
             saturate_      : 1,  // bit [ 3, 4) : 0 = no saturate. 1 = saturate. 1 value valid only for S8
             c_format_      : 2,  // bit [ 4, 6) : 0 = F16. 1 = F32, 2 = S32
                            : 1,  //
             a_format_      : 3,  // bit [ 7,10) : MXF8F6F4Format:0 = E4M3, 1 = E5M2, 3 = E2M3, 4 = E3M2, 5 = E2M1. F32F16Format: 0 = F16, 1 = BF16, 2 = TF32. S8: 0 unsigned 8 bit, 1 signed 8 bit. Boolean MMA: 0 Boolean
             b_format_      : 3,  // bit [10,13) : MXF8F6F4Format:0 = E4M3, 1 = E5M2, 3 = E2M3, 4 = E3M2, 5 = E2M1. F32F16Format: 0 = F16, 1 = BF16, 2 = TF32. S8: 0 unsigned 8 bit, 1 signed 8 bit. Boolean MMA: 0 Boolean
             a_negate_      : 1,  // bit [13,14) : 0 = no negate. 1 = negate. 1 value valid only for F32F16Format and MXF8F6F4Format
             b_negate_      : 1,  // bit [14,15) : 0 = no negate. 1 = negate. 1 value valid only for F32F16Format and MXF8F6F4Format
             a_major_       : 1;  // bit [15,16) : 0 = K-major. 1 = MN-major. Major value of 1 is only valid for E4M3, E5M2, INT8 (signed and unsigned), F16, BF16 and TF32 source formats
    uint16_t b_major_       : 1,  // bit [16,17) : 0 = K-major. 1 = MN-major. Major value of 1 is only valid for E4M3, E5M2, INT8 (signed and unsigned), F16, BF16 and TF32 source formats
             n_dim_         : 6,  // bit [17,23) : 3 LSBs not included. Valid values range from 1 (N=8) to 32 (N=256).  All values are not valid for all instruction formats
                            : 1,  //
             m_dim_         : 5,  // bit [24,29) : 4 LSBs not included. Valid values are: 4 (M=64), 8 (M=128), 16 (M=256)
                            : 1,  //
             max_shift_     : 2;  // bit [30,32) : Maximum shift for WS instruction. Encoded as follows: 0 = no shift, 1 = maximum shift of 8, 2 = maximum shift of 16, 3 = maximum shift of 32.
  };

  // Decay to a uint32_t
  HOST_DEVICE constexpr explicit
  operator uint32_t() const noexcept { return desc_; }
};
__forceinline__ __device__ void ptx_tcgen05_encode_matrix_descriptor(uint64_t* desc, void* addr, int ldo, int sdo, int swizzle) {
  SmemDescriptor _desc;

  _desc.version_ = 1;
  _desc.lbo_mode_ = 0;

  switch (swizzle) {
    case 0: _desc.layout_type_ = uint8_t(0); break; // No swizzle
    case 1: _desc.layout_type_ = uint8_t(6); break; // 32B swizzle
    case 2: _desc.layout_type_ = uint8_t(4); break; // 64B swizzle
    case 3: _desc.layout_type_ = uint8_t(2); break; // 128B swizzle
    case 4: _desc.layout_type_ = uint8_t(1); break; // 128B_base32B swizzle
  }

  uint32_t start_address = __cvta_generic_to_shared(addr);
  _desc.start_address_ = static_cast<uint16_t>(start_address >> 4);

  constexpr uint8_t base_offset = 0;
  _desc.base_offset_ = base_offset;

  _desc.stride_byte_offset_  = static_cast<uint32_t>(sdo);
  _desc.leading_byte_offset_ = static_cast<uint32_t>(ldo);

  *desc = (uint64_t)_desc;
}



__forceinline__ __device__ void ptx_tcgen05_encode_instr_descriptor(uint32_t* desc, int M, int N, int d_format,
                                            int a_format, int b_format, bool trans_a, bool trans_b,
                                            bool neg_a, bool neg_b, bool sat_d, bool is_sparse) {
  InstrDescriptor _desc;

  _desc.a_format_ = uint8_t(a_format);
  _desc.b_format_ = uint8_t(b_format);
  _desc.c_format_ = uint8_t(d_format);

  _desc.m_dim_ = (M >> 4);
  _desc.n_dim_ = (N >> 3);

  _desc.a_major_ = static_cast<uint8_t>(trans_a);
  _desc.b_major_ = static_cast<uint8_t>(trans_b);

  _desc.a_negate_ = static_cast<uint8_t>(neg_a);
  _desc.b_negate_ = static_cast<uint8_t>(neg_b);
  _desc.saturate_ = static_cast<uint8_t>(sat_d);

  _desc.sparse_flag_ = is_sparse;
  _desc.sparse_id2_  = 0;                          // should modify in sparse case

  _desc.max_shift_ = uint8_t(0);                   // WS not used

  *desc = (uint32_t)_desc;
}


extern "C" __global__ void __launch_bounds__(128) test_mma_ss_no_tma_kernel(half* __restrict__ A, half* __restrict__ B, float* __restrict__ C);
extern "C" __global__ void __launch_bounds__(128) test_mma_ss_no_tma_kernel(half* __restrict__ A, half* __restrict__ B, float* __restrict__ C) {
  __shared__ alignas(64) half A_smem[8192];
  __shared__ alignas(64) half B_smem[8192];
  alignas(64) float reg[128];
  __shared__ alignas(8) uint tmem_addr[1];
  alignas(64) uint64_t descA[1];
  alignas(64) uint64_t descB[1];
  alignas(64) uint descI[1];
  unsigned int smem_addr = __cvta_generic_to_shared(tmem_addr);
  if (((int)threadIdx.x) < 32) {
    __asm__ __volatile__(
      "tcgen05.alloc.cta_group::1.sync.aligned.shared::cta.b32 [%0], %1;"
      :: "r"(smem_addr), "r"(512)
      : "memory"
    );
  }
  __syncthreads();
  for (int i = 0; i < 128; ++i) {
    reg[i] = 0.000000e+00f;
  }
  if (((int)threadIdx.x) == 0) {
    for (int v = 0; v < 128; ++v) {
      for (int v_1 = 0; v_1 < 64; ++v_1) {
        A_smem[(((((v * 8) + (v_1 >> 3)) ^ ((((v * 8) + (v_1 >> 3)) & 56) >> 3)) << 3) + (v_1 & 7))] = A[((v * 64) + v_1)];
      }
    }
  }
  if (((int)threadIdx.x) == 0) {
    for (int v_2 = 0; v_2 < 128; ++v_2) {
      for (int v_3 = 0; v_3 < 64; ++v_3) {
        B_smem[(((((v_2 * 8) + (v_3 >> 3)) ^ ((((v_2 * 8) + (v_3 >> 3)) & 56) >> 3)) << 3) + (v_3 & 7))] = B[((v_2 * 64) + v_3)];
      }
    }
  }
  __syncthreads();
  __shared__ unsigned long long sem;
  if (((int)threadIdx.x) == 0) {
    asm volatile("mbarrier.init.shared::cta.b64 [%0], 1;" :: "l"(&sem));
    uint64_t sem_addr = static_cast<uint64_t>(__cvta_generic_to_shared(&sem));
    ptx_tcgen05_encode_instr_descriptor(descI, 128, 128, 1, 0, 0, false, false, false, false, false, false);
    for (int k = 0; k < 4; ++k) {
      ptx_tcgen05_encode_matrix_descriptor(descA, (&(A_smem[(((k * 2) ^ (((k * 2) & 56) >> 3)) << 3)])), 1, 64, 3);
      ptx_tcgen05_encode_matrix_descriptor(descB, (&(B_smem[(((k * 2) ^ (((k * 2) & 56) >> 3)) << 3)])), 1, 64, 3);
      if (k == 0) {
        
        {
            /* T.ptx_tcgen05_mma() */
            asm volatile(
                "{\n"
                ".reg .pred p;\n"
                "setp.eq.u32 p, 1, 0;\n"
                "tcgen05.mma.cta_group::1.kind::f16 [%0], %1, %2, %3, "
                "{%5, %6, %7, %8}, p;\n"
                "}\n"
                :
                : "r"(tmem_addr[0]), "l"(descA[0]), "l"(descB[0]), "r"(descI[0]), "r"(0), "r"(0), "r"(0), "r"(0), "r"(0)
            );
        }
      } else {
        
        {
            /* T.ptx_tcgen05_mma() */
            asm volatile(
                "{\n"
                ".reg .pred p;\n"
                "setp.eq.u32 p, 1, 1; \n"
                "tcgen05.mma.cta_group::1.kind::f16 [%0], %1, %2, %3, "
                "{%5, %6, %7, %8}, p;\n"
                "}\n"
                :
                : "r"(tmem_addr[0]), "l"(descA[0]), "l"(descB[0]), "r"(descI[0]), "r"(1), "r"(0), "r"(0), "r"(0), "r"(0)
            );
        }
      }
    }
    asm volatile("tcgen05.commit.cta_group::1.mbarrier::arrive::one.b64 [%0];" :: "l"(sem_addr) : "memory");
  }
  mbarrier_wait<0>(&sem);
  __syncthreads();
  for (int i_1 = 0; i_1 < 128; ++i_1) {
    
    {
        /* T.ptx_tcgen05_ld() */
        asm volatile(
            "tcgen05.ld.sync.aligned.32x32b.x1.b32 "
            "{%0}, "
            "[%1];\n"
            :  "=r"(*(uint32_t*)&reg[i_1])
            :  "r"(get_tmem_addr(tmem_addr[0], ((((int)threadIdx.x) >> 5) * 32), i_1))
        );
    }
  }
  asm volatile(
            "tcgen05.wait::ld.sync.aligned;"
        );
  for (int i_2 = 0; i_2 < 128; ++i_2) {
    C[((((int)threadIdx.x) * 128) + i_2)] = reg[i_2];
  }
  if (((int)threadIdx.x) < 32) {
        __asm__ __volatile__(
      "tcgen05.dealloc.cta_group::1.sync.aligned.b32 %0, %1;"
      :: "r"(tmem_addr[0]), "r"(512)
      : "memory"
    );
  }
}


/* ---------------- CPU reference (full matrix) ------------------ */
void cpu_gemm(const std::vector<Tab>& A,
              const std::vector<Tab>& B,
              std::vector<Tacc>& C)
{
    #pragma omp parallel for collapse(2) schedule(static)
    for (int i = 0; i < Mb; ++i) {
        for (int j = 0; j < Nb; ++j) {
            float s = 0.f;
            for (int k = 0; k < Kb; ++k)
                s += __half2float(A[i * Kb + k]) * __half2float(B[j*Kb + k]);
            C[i * Nb + j] = s;
        }
    }
}
/* =================================================================
 *                       B E N C H M A R K   D R I V E R
 * =================================================================*/

/* Forward declaration */
void gpu_run(const Tab* dA, const Tab* dB, Tacc* dC);

/* Lightning-style inner kernel launcher */
void gpu_run( Tab* dA,  Tab* dB, Tacc* dC)
{
    dim3 grid(1);
    dim3 block(128);
    test_mma_ss_no_tma_kernel<<<grid, block, 0>>>(dA, dB, dC);
}

/* Benchmark harness (matches ThunderKittens “run_benchmark”) */
int run_benchmark()
{
    std::cout
        << "--------------------  M=" << Mb
        << " N=" << Nb << " K=" << Kb << "  --------------------\n"
        << "Block size: " << Mb << "×" << Nb << " (single CTA)\n";

    /* ---------- host buffers ------------------------------------ */
    std::vector<float> hA_fp32(Mb*Kb), hB_fp32(Kb*Nb);
    std::vector<Tab>   hA(Mb*Kb),      hB(Kb*Nb);
    std::vector<Tacc>  hC_gpu(Mb*Nb, 0.f), hC_ref(Mb*Nb, 0.f);

    std::mt19937 gen(41);
    std::uniform_real_distribution<float> rnd(-1.f, 1.f);
    // for (auto& x : hA_fp32) x = rnd(gen);
    for(int i = 0; i < 128; i++){
        for(int j = 0; j < 64; j++){
            hA_fp32[i*Kb+j] = j/32;
        }
    }
    // for (auto& x : hB_fp32) x = rnd(gen);
    for(int i = 0; i < 128; i++){
        for(int j = 0; j < 64; j++){
            hB_fp32[i*Kb+j] = j/32;
        }
    }
    /* convert to fp16 */
    for (size_t i = 0; i < hA.size(); ++i) hA[i] = __float2half(hA_fp32[i]);
    for (size_t i = 0; i < hB.size(); ++i) hB[i] = __float2half(hB_fp32[i]);

    std::cout << "Host data initialised\n";

    /* ---------- reference --------------------------------------- */
    cpu_gemm(hA, hB, hC_ref);
    std::cout << "CPU reference done\n";

    /* ---------- device buffers ---------------------------------- */
    Tab  *dA, *dB;
    Tacc *dC;
    hipMalloc(&dA, hA.size()*sizeof(Tab));
    hipMalloc(&dB, hB.size()*sizeof(Tab));
    hipMalloc(&dC, hC_gpu.size()*sizeof(Tacc));
    hipMemcpy(dA, hA.data(), hA.size()*sizeof(Tab), hipMemcpyHostToDevice);
    hipMemcpy(dB, hB.data(), hB.size()*sizeof(Tab), hipMemcpyHostToDevice);
    // cudaCheckErrors();
    std::cout << "device buffer done\n";

    /* ---------- timing ------------------------------------------ */
    const int iters = (std::getenv("NCU") ? 1 : 5);
    gpu_run(dA, dB, dC);                       // 1 warm-up
    hipDeviceSynchronize();
    std::cout << "warmup done\n";

    auto t0 = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < iters; ++i) gpu_run(dA, dB, dC);
    hipDeviceSynchronize();
    auto t1 = std::chrono::high_resolution_clock::now();

    double usec = std::chrono::duration<double>(t1 - t0).count()*1e6/iters;

    /* ---------- copy back & verify ------------------------------ */
    hipMemcpy(hC_gpu.data(), dC, hC_gpu.size()*sizeof(Tacc),
               hipMemcpyDeviceToHost);
    // cudaCheckErrors();
    std::cout << "copied back, doing error checking\n";

    /* ---------- full-matrix print + error stats -------------------- */
    double max_err = 0.0, avg_err = 0.0;


    /* --- EXPECTED (CPU) ------------------------------------------- */
    std::cout << "\n=== EXPECTED (CPU) ===\n     ";
    for (int j = 0; j < Nb; ++j) std::cout << j;
    std::cout << '\n';

    for (int i = 0; i < Mb; ++i) {
        std::cout <<i << " :";
        for (int j = 0; j < Nb; ++j)
            std::cout <<  hC_ref[i * Nb + j];
        std::cout << '\n';
    }

    /* --- GPU OUTPUT ------------------------------------------------ */
    std::cout << "\n=== GPU OUTPUT ===\n     ";
    for (int j = 0; j < Nb; ++j) std::cout <<  j;
    std::cout << '\n';

    for (int i = 0; i < Mb; ++i) {
        std::cout <<  i << " :";
        for (int j = 0; j < Nb; ++j) {
            float gpu = hC_gpu[i * Nb + j];
            std::cout <<  gpu;

            /* accumulate error stats */
            double e = std::fabs(gpu - hC_ref[i * Nb + j]);
            max_err  = std::max(max_err, e);
            avg_err += e;
        }
        std::cout << '\n';
    }

    
    std::ofstream fout("matrix_dump.txt");
    std::streambuf* cout_buf = std::cout.rdbuf(); // backup
    std::cout.rdbuf(fout.rdbuf());                // redirect


    std::cout << "\n=== EXPECTED (CPU) ===\n     ";
    for (int j = 0; j < Nb; ++j) std::cout<< j;
    std::cout << '\n';

    for (int i = 0; i < Mb; ++i) {
        std::cout  << i << " :";
        for (int j = 0; j < Nb; ++j)
            std::cout << hC_ref[i * Nb + j];
        std::cout << '\n';
    }

    std::cout << "\n=== GPU OUTPUT ===\n     ";
    for (int j = 0; j < Nb; ++j) std::cout << j;
    std::cout << '\n';

    for (int i = 0; i < Mb; ++i) {
        std::cout  << i << " :";
        for (int j = 0; j < Nb; ++j) {
            float gpu = hC_gpu[i * Nb + j];
            std::cout << gpu;

            double e = std::fabs(gpu - hC_ref[i * Nb + j]);
            max_err  = std::max(max_err, e);
            avg_err += e;
        }
        std::cout << '\n';
    }

    avg_err /= static_cast<double>(Mb * Nb);
    std::cout << "\nSummary  →  max|err| = " << max_err
              << "   avg|err| = " << avg_err << '\n';

    std::cout.rdbuf(cout_buf); // restore std::cout
    /* ---------- clean up ---------------------------------------- */
    hipFree(dA); hipFree(dB); hipFree(dC);
    return 0;

}

/* ------------------------- main ---------------------------------- */
int main()
{
    /* optional: force CUDA context creation upfront */
    hipFree(nullptr);

    run_benchmark();
    return 0;
}